#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <chrono>
#include <cstdio>

using namespace std::chrono;

__global__ void work(const char *buffer, size_t *tokens, int *token_index) {

  size_t i = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + 1;

  if (buffer[i] == '\r') {
    if (buffer[i + 1] == '\n') {
      tokens[atomicAdd(token_index, 1)] = i + 2;
    }
  } else if (buffer[i] == '\n') {
    if (buffer[i - 1] == '\r') {
      tokens[atomicAdd(token_index, 1)] = i + 1;
    }
  }
}

void *allocHostMemory(size_t size) {
  char *buffer;
  checkCudaErrors(hipHostMalloc(&buffer, size));
  return buffer;
}

void freeHostMemory(void *ptr) { checkCudaErrors(hipHostFree(ptr)); }

void test(const char *buffer, size_t buffer_size, size_t *breaks,
          size_t max_breaks_count, steady_clock::time_point &start,
          steady_clock::time_point &stop) {

  auto t1 = steady_clock::now();

  int total_gpu_count;
  checkCudaErrors(hipGetDeviceCount(&total_gpu_count));

  int master_device = 0;
  const size_t use_gpu_count = 4;

  printf(
      "Device count is %d, using first %zu devices, the master device is %d\n",
      total_gpu_count, use_gpu_count, master_device);

  for (size_t i = 0; i < use_gpu_count; i++) {
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipFree(0));
    if (i > 0) {
      checkCudaErrors(hipDeviceEnablePeerAccess(0, 0));
    }
  }

  const char *d_buffer[use_gpu_count];
  size_t *d_breaks;
  int *d_break_index;
  size_t d_buffer_padding_size =
      (buffer_size % 2048 != 0) ? (2048 - buffer_size % 2048) : 0;

  for (size_t i = 0; i < use_gpu_count; i++) {
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipMalloc((void **)&d_buffer[i],
                               i == 0 ? buffer_size + d_buffer_padding_size + 1
                                      : (buffer_size + d_buffer_padding_size) /
                                            use_gpu_count));
  }

  checkCudaErrors(hipSetDevice(master_device));
  checkCudaErrors(
      hipMalloc((void **)&d_breaks, max_breaks_count * sizeof(size_t)));
  checkCudaErrors(hipMalloc((void **)&d_break_index, sizeof(int)));

  auto t2 = steady_clock::now();

  for (size_t i = 0; i < use_gpu_count; i++) {
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipMemcpyAsync(
        (void *)d_buffer[i],
        buffer + i * (buffer_size + d_buffer_padding_size) / use_gpu_count,
        (buffer_size + d_buffer_padding_size) / use_gpu_count,
        hipMemcpyHostToDevice));
  }

  checkCudaErrors(hipSetDevice(master_device));
  checkCudaErrors(hipMemsetAsync((void *)(d_buffer[0] + buffer_size), 0,
                                  d_buffer_padding_size + 1));
  checkCudaErrors(hipMemsetAsync(d_break_index, 0, sizeof(int)));

  for (size_t i = 1; i < use_gpu_count; i++) {
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipMemcpyPeerAsync(
        (void *)(d_buffer[0] +
                 i * (buffer_size + d_buffer_padding_size) / use_gpu_count),
        0, d_buffer[i], i,
        (buffer_size + d_buffer_padding_size) / use_gpu_count));
  }
  for (size_t i = 1; i < use_gpu_count; i++) {
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipDeviceSynchronize());
  }

  auto t3 = steady_clock::now();

  size_t threads = (buffer_size + d_buffer_padding_size) / 2;
  const size_t block_dim = 1024;
  size_t blocks = threads / block_dim;
  printf("Run kernel, blocks=%zu, block_dim=%zu\n", blocks, block_dim);

  checkCudaErrors(hipSetDevice(master_device));
  work<<<blocks, block_dim>>>(d_buffer[0], d_breaks, d_break_index);
  checkCudaErrors(hipDeviceSynchronize());

  auto t4 = steady_clock::now();

  checkCudaErrors(hipMemcpy(breaks, d_breaks,
                             max_breaks_count * sizeof(size_t),
                             hipMemcpyDeviceToHost));

  for (size_t i = 0; i < use_gpu_count; i++) {
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipFree((void *)d_buffer[i]));
  }
  checkCudaErrors(hipSetDevice(master_device));
  checkCudaErrors(hipFree((void *)d_breaks));
  checkCudaErrors(hipFree((void *)d_break_index));
  for (size_t i = 0; i < use_gpu_count; i++) {
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipDeviceSynchronize());
  }

  auto t5 = steady_clock::now();

  printf("CUDA initialization and memory allocation took %.3f milliseconds\n",
         1.0f * duration_cast<microseconds>(t2 - t1).count() / 1000);
  printf("H2D & P2P memory copy took %.3f milliseconds\n",
         1.0f * duration_cast<microseconds>(t3 - t2).count() / 1000);
  printf("Kernel took %.3f milliseconds\n",
         1.0f * duration_cast<microseconds>(t4 - t3).count() / 1000);
  printf("D2H memory copy and memory deallocation took %.3f milliseconds\n",
         1.0f * duration_cast<microseconds>(t5 - t4).count() / 1000);

  start = t1;
  stop = t5;
}