
#include <hip/hip_runtime.h>
#include <cstdio>

static const size_t BLOCK_SIZE = 128;

char *allocBuffer(size_t size)
{
    char *buffer;
    hipHostAlloc(&buffer, size, hipHostMallocMapped);
    return buffer;
}

void freeBuffer(char *buffer)
{
    hipHostFree(buffer);
}

__global__ void foo(const char *array, size_t *tokens, int *token_index)
{
    __shared__ char s_array[BLOCK_SIZE + 1];

    size_t offset = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.x == BLOCK_SIZE - 1)
    {
        s_array[threadIdx.x] = array[offset];
        s_array[threadIdx.x + 1] = array[offset + 1];
    }
    else
    {
        s_array[threadIdx.x] = array[offset];
    }
    __syncthreads();

    if (s_array[threadIdx.x] == '\r' && s_array[threadIdx.x + 1] == '\n')
    {
        int index = atomicAdd(token_index, 1);
        tokens[index] = offset + 2;
    }
}

void tokenize(const char *buffer, size_t buffer_size, size_t *tokens, size_t token_size)
{
    const char *d_buffer;
    size_t *d_tokens;
    int *d_token_index;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void **)&d_buffer, buffer_size + 1);
    hipMalloc((void **)&d_tokens, token_size * sizeof(size_t));
    hipMalloc((void **)&d_token_index, sizeof(int));
    hipMemcpy((void *)d_buffer, buffer, buffer_size, hipMemcpyHostToDevice);
    hipMemset(d_token_index, 0, sizeof(int));

    size_t blocks = buffer_size / BLOCK_SIZE;

    hipEventRecord(start);
    foo<<<blocks, BLOCK_SIZE>>>(d_buffer, d_tokens, d_token_index);
    hipEventRecord(stop);

    hipMemcpy(tokens, d_tokens, token_size * sizeof(size_t), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel took %.4f milliseconds\n", milliseconds);

    hipFree((void *)d_buffer);
    hipFree((void *)d_tokens);
    hipFree((void *)d_token_index);
}