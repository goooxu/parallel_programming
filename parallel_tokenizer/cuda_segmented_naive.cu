
#include <hip/hip_runtime.h>
#include <cstdio>

static const size_t BLOCK_SIZE = 128;
static const size_t SEGMENT_SIZE = sizeof(int);

char *allocBuffer(size_t size)
{
    char *buffer;
    hipHostAlloc(&buffer, size, hipHostMallocMapped);
    return buffer;
}

void freeBuffer(char *buffer)
{
    hipHostFree(buffer);
}

__global__ void foo(const char *array, size_t *tokens, int *token_index)
{
    __shared__ char s_array[SEGMENT_SIZE * BLOCK_SIZE + 1];

    size_t offset = (blockIdx.x * blockDim.x + threadIdx.x) * SEGMENT_SIZE;
    size_t s_offset = threadIdx.x * SEGMENT_SIZE;
    *reinterpret_cast<int *>(&s_array[s_offset]) = *reinterpret_cast<const int *>(&array[offset]);
    if (threadIdx.x == BLOCK_SIZE - 1)
        s_array[s_offset + SEGMENT_SIZE] = array[offset + SEGMENT_SIZE];
    __syncthreads();

    for (size_t i = 0; i < SEGMENT_SIZE; i++)
    {
        if (s_array[s_offset + i] == '\r' && s_array[s_offset + i + 1] == '\n')
        {
            int index = atomicAdd(token_index, 1);
            tokens[index] = offset + i + 2;
        }
    }
}

void tokenize(const char *buffer, size_t buffer_size, size_t *tokens, size_t token_size)
{
    const char *d_buffer;
    size_t *d_tokens;
    int *d_token_index;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void **)&d_buffer, buffer_size + 1);
    hipMalloc((void **)&d_tokens, token_size * sizeof(size_t));
    hipMalloc((void **)&d_token_index, sizeof(int));
    hipMemcpy((void *)d_buffer, buffer, buffer_size, hipMemcpyHostToDevice);
    hipMemset(d_token_index, 0, sizeof(int));

    size_t blocks = buffer_size / BLOCK_SIZE / SEGMENT_SIZE;

    hipEventRecord(start);
    foo<<<blocks, BLOCK_SIZE>>>(d_buffer, d_tokens, d_token_index);
    hipEventRecord(stop);

    hipMemcpy(tokens, d_tokens, token_size * sizeof(size_t), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Kernel took %.4f milliseconds\n", milliseconds);

    hipFree((void *)d_buffer);
    hipFree((void *)d_tokens);
    hipFree((void *)d_token_index);
}