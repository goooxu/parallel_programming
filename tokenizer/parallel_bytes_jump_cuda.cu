
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void foo(const char *array, const char *base, const char **tokens, int *index)
{
    int sourceIdx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;

    if (array[sourceIdx] == '\r')
    {
        if (array[sourceIdx + 1] == '\n')
        {
            int targetIdx = atomicAdd(index, 1);
            tokens[targetIdx] = base + sourceIdx + 2;
        }
    }
    else if (array[sourceIdx] == '\n')
    {
        if (sourceIdx > 0 && array[sourceIdx - 1] == '\r')
        {
            int targetIdx = atomicAdd(index, 1);
            tokens[targetIdx] = base + sourceIdx + 1;
        }
    }
}

void tokenize(const char *begin, const char *end, const char **tokens, size_t max_tokens)
{
    const char *d_array;
    const char **d_tokens;
    int *d_index;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void **)&d_array, end - begin);
    hipMalloc((void **)&d_tokens, max_tokens * sizeof(const char *));
    hipMalloc((void **)&d_index, sizeof(int));
    hipMemcpy((void *)d_array, begin, end - begin, hipMemcpyHostToDevice);
    hipMemset(d_index, 0, sizeof(int));

    int blocks = (end - begin) / 2 / 1024;

    hipEventRecord(start);
    foo<<<blocks, 1024>>>(d_array, begin, d_tokens, d_index);
    hipEventRecord(stop);

    hipMemcpy(tokens, d_tokens, max_tokens * sizeof(const char *), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Kernel took milliseconds: %.4fms\n", milliseconds);
}